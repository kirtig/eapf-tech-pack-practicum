/*  calculating pi via area under the curve
 *  This code uses an algorithm fairly easily ported to all parallel methods.
 *  Since it calculates pi, it is easy to verify that results are correct.
 *  It can also be used to explore accuracy of results and techniques for managing error.
 */


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define NUMRECT 10000000


/*  students learn in grammar school that the area of a circle is pi*radius*radius.
 *  They learn in high school that the formula of a circle is x^2 + y^2 = radius^2.
 *
 *  These facts allows students calculating pi by estimating area of mid-point rectangles
 *
 *  Area of unit circle is pi, y = sqrt(1-x^2) is formula for semicircle from -1 to 1
 */


// constants useful to CUDA
 const int        threadsPerBlock        = 256; 
 const int        blocksPerGrid                =  32;
 const int        totalThreads                = threadsPerBlock * blocksPerGrid;
 const float        overallWidth                 = 2.0f; 
 const float        block_width                 = overallWidth / blocksPerGrid;


  __global__ void calcArea(int *d_rectPerThread, float *d_width, float *partPiByBlock) {
        __shared__ float partPiByThread[threadsPerBlock];
        int reduce_i = blockDim.x / 2;        // index for reducing thread results to single block value
        float width = *d_width;
        int rectPerThread = *d_rectPerThread;
        float x                = -1.0f 
+  (overallWidth * blockIdx.x) / blocksPerGrid
+  (block_width * threadIdx.x) / threadsPerBlock
-  width / 2;
        float partPi         = 0.0f;
        
            for (int i = 0; i < rectPerThread; i++) {
                x += width;
                partPi += width * sqrtf(1.0f - x * x);
            }
            partPiByThread[threadIdx.x] = partPi;


        // reduce all threads in the block to a single block value
        while (reduce_i != 0) {
                __syncthreads();
                if (threadIdx.x < reduce_i)
                        partPiByThread[threadIdx.x] += partPiByThread[threadIdx.x + reduce_i]; 
                reduce_i /= 2;
        }
        
        // store block result in correct spot for reducing on CPU side
        if (threadIdx.x == 0) 
                partPiByBlock[blockIdx.x] = partPiByThread[0];
 }




int main(int argc, char **argv) {


        int        numRect;                                        // number of rectangles
        int *d_rectPerThread, rectPerThread;                        // number of rectangles per thread
        int        i;                                                // loop index
        float        *d_width, width;                                // width of each rectangle
        float   *d_partPiByBlock, h_partPiByBlock[blocksPerGrid]; // partial pi values returned by CUDA
        float        pi, halfPI = 0.0;                                // sum of area of rectangles gives pi/2


        numRect = argc == 2 ? atoi(argv[1]) : NUMRECT;        // get number of rectangles
        rectPerThread = numRect / totalThreads;
        numRect = rectPerThread * totalThreads;
        width = overallWidth / numRect;                        // calculate width of each rectangle


        hipMalloc((void**)&d_rectPerThread, sizeof(int));
        hipMalloc((void**)&d_width, sizeof(int));
        hipMalloc((void**)&d_partPiByBlock, sizeof(float) * blocksPerGrid); 
                
        hipMemcpy(d_rectPerThread, &rectPerThread, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_width, &width, sizeof(float), hipMemcpyHostToDevice);
        
                calcArea <<<blocksPerGrid, threadsPerBlock>>> (d_rectPerThread, d_width, d_partPiByBlock);
        
        hipMemcpy(h_partPiByBlock, d_partPiByBlock, sizeof(float) * blocksPerGrid, 
hipMemcpyDeviceToHost);


        for (i = 0; i < blocksPerGrid; ++i) 
                halfPI += h_partPiByBlock[i];        /* calculate pi/4, with room for better error mgmt */
        pi = 2.0 * halfPI;


        printf ("\n==\n==\t%20s = %15.10f\n",                "pi",                        pi);
        printf ("==\t%20s = %15d\n",                        "total rectangles",         numRect);
        printf ("==\t%20s = %15d\n==\n\n",                "CUDA threads",        totalThreads);
        return 0;
}